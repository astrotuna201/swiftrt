#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include <assert.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "compareOps.h"

//==============================================================================
// ops
//==============================================================================

template<typename E>
struct Abs {
    __device__ inline static E op(const E& x) { return abs(x); }
};

//==============================================================================
// kernels
//==============================================================================

//==============================================================================
// dynamic dispatch functions
//==============================================================================


//==============================================================================
// Swift importable C interface functions
//==============================================================================

hipError_t srtAnd(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtElementsAlmostEqual(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    const void* tolerance,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtEqual(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtGreater(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtGreaterOrEqual(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLess(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtLessOrEqual(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtMax(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtMin(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtNotEqual(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtOr(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}

hipError_t srtReplace(
    const void* a, const srtTensorDescriptor* aDesc,
    const void* b, const srtTensorDescriptor* bDesc,
    const void* condition, const srtTensorDescriptor* cDesc,
    void* out, const srtTensorDescriptor* oDesc,
    hipStream_t stream)
{
    return hipErrorNotSupported;
}
