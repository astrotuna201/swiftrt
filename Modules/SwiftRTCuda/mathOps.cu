#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "dispatchHelpers.h"
#include "mathOps.h"
#include "mathSupplemental.h"
#include <type_traits>

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//------------------------------------------------------------------------------
Op1(Abs, abs, (isSame<T,Out>() && isNumeric<T>()) || isComplex<T>())

hipError_t srtAbs(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Abs>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Acos, acos, (isSame<T,Out>() && isFloating<T>()))

// Must be promoted types
hipError_t srtAcos(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Acos>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Acosh, acosh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtAcosh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Acosh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Add, add, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtAdd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Add>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
hipError_t srtAddTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Add>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Asin, asin, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtAsin(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Asin>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Asinh, asinh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtAsinh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Asinh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Atan, atan, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtAtan(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Atan>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
hipError_t srtAtan2(
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    // Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    // // b comes first
    // return select<Atan2>(b, bDesc, a, aDesc, out, oDesc, stream);
    return hipErrorNotSupported;
}

//------------------------------------------------------------------------------
Op1(Atanh, atanh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtAtanh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Atanh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Cos, cos, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtCos(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Cos>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Cosh, cosh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtCosh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Cosh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Div, divide, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtDiv(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Div>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtDivTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Div>(a, aDesc, element, out, oDesc, stream);
}

hipError_t srtDivET(
    const void* element,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Div>(element, a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Erf, erf, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtErf(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Erf>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Erfc, erfc, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtErfc(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Erfc>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp, exp, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtExp(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp2, exp2, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtExp2(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp2>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp10, exp10, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtExp10(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp10>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(ExpMinusOne, expm1, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtExpMinusOne(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<ExpMinusOne>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Gamma, tgamma, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtGamma(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Gamma>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Hypot, hypot, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtHypot(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Hypot>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log, log, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtLog(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(LogOnePlus, log1p, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtLogOnePlus(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<LogOnePlus>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log2, log2, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtLog2(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log2>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log10, log10, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtLog10(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log10>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(LogGamma, lgamma, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtLogGamma(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<LogGamma>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Mul, multiply, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtMul(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Mul>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtMulTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Mul>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
OpTTU(MultiplyAdd, multiplyAdd, (isSame<T,Out>() && isSame<T,U>() && isNumeric<T>()))

hipError_t srtMultiplyAdd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsABC(paDesc, pbDesc, pcDesc, poDesc)
    return select<MultiplyAdd>(a, aDesc, b, bDesc, c, cDesc, out, oDesc, stream);
}

Op3(MultiplyAddE, multiplyAdd, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtMultiplyAddTTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<MultiplyAddE>(a, aDesc, b, bDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Neg, neg, (isSame<T,Out>() && (isSignedNumeric<T>() || isComplex<T>())))

hipError_t srtNeg(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Neg>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Pow, pow, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtPow(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Pow>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
hipError_t srtPowN(
    const void* a, const srtTensorDescriptor* paDesc, long n,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    // Cast2TensorDescriptorsA(paDesc, poDesc)
    // return select<PowN>(a, aDesc, int(n), out, oDesc, stream);
    return hipErrorNotSupported;
}

//------------------------------------------------------------------------------
hipError_t srtRoot(
    const void* a, const srtTensorDescriptor* paDesc, long n,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    // Cast2TensorDescriptorsA(paDesc, poDesc)
    // return select<Root>(a, aDesc, int(n), out, oDesc, stream);
    return hipErrorNotSupported;
}

//------------------------------------------------------------------------------
Op1(Sigmoid, sigmoid, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtSigmoid(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sigmoid>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sign, sign, (isSame<T,Out>() && isSignedNumeric<T>()))

hipError_t srtSign(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sign>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sin, sin, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtSin(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sin>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sinh, sinh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtSinh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sinh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sqrt, sqrt, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtSqrt(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sqrt>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Squared, squared, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtSquared(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream)
{
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Squared>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Sub, subtract, (isSame<T,Out>() && isNumeric<T>()))

hipError_t srtSub(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Sub>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtSubTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sub>(a, aDesc, element, out, oDesc, stream);
}

hipError_t srtSubET(
    const void* element,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sub>(element, a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Tan, tan, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtTan(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Tan>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Tanh, tanh, (isSame<T,Out>() && isFloating<T>()))

hipError_t srtTanh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Tanh>(a, aDesc, out, oDesc, stream);
}