#include "hip/hip_runtime.h"
//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "math_fn.h"
#include "op1.h"
#include "op2.h"
#include "op3.h"
#include "srt_types.h"

//==============================================================================
// Swift importable C interface functions
//==============================================================================

//------------------------------------------------------------------------------
Op1(Abs, abs, (isNumeric<A>() && (isSame<A,Out>() || isComplex<A>())))

hipError_t srtAbs(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectT_O<Abs>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Acos, acos, isFloating<A>())

// Must be promoted types
hipError_t srtAcos(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Acos>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Acosh, acosh, isFloating<A>())

hipError_t srtAcosh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Acosh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Add, add, isNumeric<A>())

hipError_t srtAdd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Add>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
hipError_t srtAddTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Add>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Asin, asin, isFloating<A>())

hipError_t srtAsin(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Asin>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Asinh, asinh, isFloating<A>())

hipError_t srtAsinh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Asinh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Atan, atan, isFloating<A>())

hipError_t srtAtan(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Atan>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Atan2, atan2, isFloating<A>())

hipError_t srtAtan2(
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    // b comes first
    return select<Atan2>(b, bDesc, a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Atanh, atanh, isFloating<A>())

hipError_t srtAtanh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Atanh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Cos, cos, isFloating<A>())

hipError_t srtCos(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Cos>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Cosh, cosh, isFloating<A>())

hipError_t srtCosh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Cosh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Div, divide, isNumeric<A>())

hipError_t srtDiv(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Div>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtDivTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Div>(a, aDesc, element, out, oDesc, stream);
}

// `true` swaps `a` and `element` when calling `divide`
Op2SwapAB(DivET, divide, isNumeric<A>())

hipError_t srtDivET(
    const void* element,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<DivET>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Erf, erf, isFloating<A>())

hipError_t srtErf(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Erf>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Erfc, erfc, isFloating<A>())

hipError_t srtErfc(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Erfc>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp, exp, isFloating<A>())

hipError_t srtExp(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp2, exp2, isFloating<A>())

hipError_t srtExp2(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp2>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Exp10, exp10, isFloating<A>())

hipError_t srtExp10(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Exp10>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(ExpMinusOne, expm1, isFloating<A>())

hipError_t srtExpMinusOne(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<ExpMinusOne>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Gamma, tgamma, isFloating<A>())

hipError_t srtGamma(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Gamma>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Hypot, hypot, isFloating<A>())

hipError_t srtHypot(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Hypot>(a, aDesc, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log, log, isFloating<A>())

hipError_t srtLog(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(LogOnePlus, log1p, isFloating<A>())

hipError_t srtLogOnePlus(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<LogOnePlus>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log2, log2, isFloating<A>())

hipError_t srtLog2(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log2>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Log10, log10, isFloating<A>())

hipError_t srtLog10(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Log10>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(LogGamma, lgamma, isFloating<A>())

hipError_t srtLogGamma(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<LogGamma>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Mul, multiply, isNumeric<A>())

hipError_t srtMul(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Mul>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtMulTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Mul>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op3(MultiplyAdd, multiplyAdd, isNumeric<A>())

hipError_t srtMultiplyAdd(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* c, const srtTensorDescriptor* pcDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsABC(paDesc, pbDesc, pcDesc, poDesc)
    return select<MultiplyAdd>(a, aDesc, b, bDesc, c, cDesc, out, oDesc, stream);
}

Op3SwapBC(MultiplyAddE, multiplyAdd, isNumeric<A>())

hipError_t srtMultiplyAddTTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<MultiplyAddE>(a, aDesc, element, b, bDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Neg, neg, (isSignedNumeric<A>() || isComplex<A>()))

hipError_t srtNeg(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Neg>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Pow, pow, isFloating<A>())

hipError_t srtPow(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Pow>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtPowTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* exponent,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Pow>(a, aDesc, exponent, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sigmoid, sigmoid, isFloating<A>())

hipError_t srtSigmoid(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sigmoid>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sign, sign, isSignedNumeric<A>())

hipError_t srtSign(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sign>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sin, sin, isFloating<A>())

hipError_t srtSin(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sin>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sinh, sinh, isFloating<A>())

hipError_t srtSinh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sinh>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Sqrt, sqrt, isFloating<A>())

hipError_t srtSqrt(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sqrt>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Squared, squared, isNumeric<A>())

hipError_t srtSquared(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Squared>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op2(Sub, subtract, isNumeric<A>())

hipError_t srtSub(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* b, const srtTensorDescriptor* pbDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsAB(paDesc, pbDesc, poDesc)
    return select<Sub>(a, aDesc, b, bDesc, out, oDesc, stream);
}

hipError_t srtSubTE(
    const void* a, const srtTensorDescriptor* paDesc,
    const void* element,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Sub>(a, aDesc, element, out, oDesc, stream);
}

// `true` swaps `a` and `element` when calling `divide`
Op2SwapAB(SubET, subtract, isNumeric<A>())

hipError_t srtSubET(
    const void* element,
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<SubET>(a, aDesc, element, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Tan, tan, isFloating<A>())

hipError_t srtTan(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Tan>(a, aDesc, out, oDesc, stream);
}

//------------------------------------------------------------------------------
Op1(Tanh, tanh, isFloating<A>())

hipError_t srtTanh(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return select<Tanh>(a, aDesc, out, oDesc, stream);
}