//******************************************************************************
// Copyright 2019 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
#include "reduce_fn.cuh"
#include "op1.h"

using namespace hipcub;

//==============================================================================
// Swift importable C interface functions
//==============================================================================

hipError_t srtAbsSum(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectType<AbsSumOp>(a, aDesc, out, oDesc, stream);
}

hipError_t srtAll(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    assert(aDesc.type == boolean && oDesc.type == boolean);
    return reduce<AllOp, bool>(a, aDesc, out, oDesc, stream);
}

hipError_t srtAny(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    assert(aDesc.type == boolean && oDesc.type == boolean);
    return reduce<AnyOp, bool>(a, aDesc, out, oDesc, stream);
}

hipError_t srtSum(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectType<SumOp>(a, aDesc, out, oDesc, stream);
}

hipError_t srtMean(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtMinValue(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectType<MinOp>(a, aDesc, out, oDesc, stream);
}

hipError_t srtArgMin(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectType<SumOp>(a, aDesc, out, oDesc, stream);
}

hipError_t srtMaxValue(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    Cast2TensorDescriptorsA(paDesc, poDesc)
    return selectType<MaxOp>(a, aDesc, out, oDesc, stream);
}

hipError_t srtArgMax(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtProd(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}

hipError_t srtProdNonZeros(
    const void* a, const srtTensorDescriptor* paDesc,
    void* out, const srtTensorDescriptor* poDesc,
    hipStream_t stream
) {
    return hipErrorNotSupported;
}
